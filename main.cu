
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>

#define N 512

__global__ void add(int *a, int *b, int *c){
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void randomInts(int *arr, int size) {
	for (int i = 0; i < size; i++) {
		arr[i] = rand();
	}
}

int main(void) {
	srand(time(NULL));

	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = (int *)malloc(size); randomInts(a, N);
	b = (int *)malloc(size); randomInts(b, N);
	c = (int *)malloc(size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	add<<<N,1>>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	
	printf("calculated!\n");

	for (int i = 0; i < N; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

 	free(a); free(b); free(c);	
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
			
