#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements) {
		C[i] = A[i] + B[i] + 0.0f;
	}
}


int main(void) {
	hipError_t err = hipSuccess;
	int numElements = 50000;
	size_t size = numElements * sizeof(float);
	printf("[Vector Addition of %d elements]\n", numElements);

	float *h_A = (float *) malloc(size);
	float *h_B = (float *) malloc(size);
	float *h_C = (float *) malloc(size);

	if ((h_A == NULL) || (h_B == NULL) || (h_C == NULL)) {
		fprintf(stderr, "Failde to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	for (int i = 0; i < numElements; ++i) {
		h_A[i] = rand() / (float)RAND_MAX;
		h_B[i] = rand() / (float)RAND_MAX;
	}

	float *d_A = NULL;
	err = hipMalloc((void **)&d_A, size);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
		hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	float *d_B = NULL;
	err = hipMalloc((void **)&d_B, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",
		hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	float *d_C = NULL;
	err = hipMalloc((void **)&d_C, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
		hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("Copy input data from the host memory to the CUDA device\n");
  	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n",hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
    	exit(EXIT_FAILURE);
  	}
	
	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  	if (err != hipSuccess) {
    	fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
    	exit(EXIT_FAILURE);
  	}

	for (int i = 0; i < numElements; ++i) {
		if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			exit(EXIT_FAILURE);
		}
	}
	printf("Test PASSED\n");
	
	err = hipFree(d_A);
	if (err != hipSuccess) {
	  	fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
	  	exit(EXIT_FAILURE);
	}
  
	err = hipFree(d_B);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
	  	exit(EXIT_FAILURE);
	}
  
	err = hipFree(d_C);
	if (err != hipSuccess) {
	  	fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
	  	exit(EXIT_FAILURE);
	}
  
	// Free host memory
	free(h_A);
	free(h_B);
	free(h_C);
  
	printf("Done\n");
	return 0;
}
			
