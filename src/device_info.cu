#include <stdio.h>
#include "hip/hip_runtime.h"

int main(void) {
	int count = 0;
	hipDeviceProp_t prop;

	hipGetDeviceCount(&count);

	printf("Count of CUDA device(s): %d\n", count);

	for (int i = 0; i < count; i++ ) {
		hipGetDeviceProperties(&prop, i);
		printf(" --- General inforamation about device #%d --- \n", i);
		printf("Name: %s\n", prop.name);
		printf("Version: %d.%d\n", prop.major, prop.minor);
		printf("Clock rate: %d\n", prop.clockRate);
		printf("Device Overlap: %s\n", prop.deviceOverlap ? "Available" : "Disable");
		printf("Wrap size: %d\n", prop.warpSize);
		printf("Threads per block: %d\n", prop.maxThreadsPerBlock);
		printf("Max THREAD size: %d x %d x %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max GRID size: %d x %d x %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	}

	return 0;
}
